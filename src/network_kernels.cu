#include "dark_cuda.h"

#include <stdio.h>
#include <time.h>
#include <assert.h>
#include <unistd.h>
#include <signal.h>
#include <sys/resource.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"
#include "parser.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "rnn_layer.h"
#include "gru_layer.h"
#include "crnn_layer.h"
#include "detection_layer.h"
#include "region_layer.h"
#include "convolutional_layer.h"
#include "activation_layer.h"
#include "maxpool_layer.h"
#include "reorg_layer.h"
#include "avgpool_layer.h"
#include "normalization_layer.h"
#include "batchnorm_layer.h"
#include "cost_layer.h"
#include "local_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
#include "route_layer.h"
#include "shortcut_layer.h"
#include "blas.h"

//#ifdef OPENCV
//#include <opencv2/highgui/highgui_c.h>
//#endif

#include "http_stream.h"

float * get_network_output_gpu_layer(network net, int i);
float * get_network_delta_gpu_layer(network net, int i);
float * get_network_output_gpu(network net);
void enqueue(int* q, int val);
int dequeue(int* q);

extern int* test_extern_arr;
extern int identifier;
extern int * queue;
extern pthread_mutex_t *gpu_lock;
extern int N;

void forward_network_gpu(network net, network_state state)
{
    hipDeviceSynchronize();
    //printf("\n");
    state.workspace = net.workspace;
    state.workspace_cpu = net.workspace_cpu;
    int i;
    int *res_arr;
    double _time;
    double time;
    res_arr = test_extern_arr;
    for(i = 0; i < net.n; ++i){
        
        state.index = i;
        layer l = net.layers[i];
        
        if(l.delta_gpu && state.train){
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }   
        
        time  = get_time_point();
        
        if (res_arr[i] == 0){ // on cpu
            if (l.type == CONVOLUTIONAL && net.quantized == 1 && l.index >=1 && l.activation != LINEAR) {
                l.forward_quant(l, state); // w/ quantize
            }
            else {
                l.forward(l,state);   //  w/o quantize
            }
        }
        else{ // on gpu 
            
            // gpu access control by mutex
            while(pthread_mutex_trylock(gpu_lock)){
                //printf("[Process %d put into wait]\n", identifier);
                enqueue(queue, getpid());
                kill(getpid(), SIGSTOP);
                setpriority(PRIO_PROCESS, getpid(), -20);
                continue;
            }
            
            l.forward_gpu(l, state);
            CHECK_CUDA(hipDeviceSynchronize());
        }
        setpriority(PRIO_PROCESS, getpid(), -10-identifier);
        //printf("[Process %d] layer: %3d type: %15s - Predicted in %8.5f milli-seconds.\n", identifier, i, get_layer_string(l.type), ((double)get_time_point() -time) / 1000);
        
        pthread_mutex_unlock(gpu_lock);
        kill( dequeue(queue), SIGCONT);

        if(net.wait_stream)
            hipStreamSynchronize(get_cuda_stream());

        if(res_arr[i] == 0){//currently running on CPU
            if(res_arr[i+1] == 0){//next is running on CPU
                state.input = l.output;    
            }
            else{//next is running on GPU
                _time = get_time_point();
                //cuda_push_array(l.output_gpu, l.output, l.batch*l.outputs);
                state.input = l.output;
            }
        }
        else{//currently running on GPU
            if(res_arr[i+1] == 0){//next is running on CPU
                _time = get_time_point();
                //cuda_pull_array(l.output_gpu, l.output, l.batch*l.outputs);
                state.input = l.output_gpu;
            }
            else{//next is running on GPU
                state.input = l.output_gpu;
            }
        }
    }
}

void backward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    float * original_input = state.input;
    float * original_delta = state.delta;
    for(i = net.n-1; i >= 0; --i){
        state.index = i;
        layer l = net.layers[i];
        if (l.stopbackward) break;
        if(i == 0){
            state.input = original_input;
            state.delta = original_delta;
        }else{
            layer prev = net.layers[i-1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
        }
        if (l.onlyforward) continue;
        l.backward_gpu(l, state);

        /*
        if(i != 0)
        {
            layer l = net.layers[i - 1];
            int state_delta_nan_inf = is_nan_or_inf(state.delta, l.outputs * l.batch);
            int state_input_nan_inf = is_nan_or_inf(state.input, l.outputs * l.batch);
            printf("\n i - %d  is_nan_or_inf(s.delta) = %d \n", i, state_delta_nan_inf);
            printf(" i - %d  is_nan_or_inf(s.input) = %d \n", i, state_input_nan_inf);
            if (state_delta_nan_inf || state_input_nan_inf) { printf(" found "); getchar(); }
        }
        */
    }
}

void update_network_gpu(network net)
{
    cuda_set_device(net.gpu_index);
    int i;
    int update_batch = net.batch*net.subdivisions * get_sequence_value(net);
    float rate = get_current_rate(net);
    for(i = 0; i < net.n; ++i){
        layer l = net.layers[i];
        l.t = get_current_batch(net);
        if(l.update_gpu){
            l.update_gpu(l, update_batch, rate, net.momentum, net.decay);
        }
    }
}

void forward_backward_network_gpu(network net, float *x, float *y)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(net.layers[net.n-1].truths) y_size = net.layers[net.n-1].truths*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    state.truth = *net.truth_gpu;
    state.train = 1;
#if defined(CUDNN_HALF) && defined(CUDNN)
    int i;
    for (i = 0; i < net.n; ++i) {
        layer l = net.layers[i];
        if (net.cudnn_half){
            if (l.type == CONVOLUTIONAL && l.weights_gpu && l.weights_gpu16) {
                assert((l.nweights) > 0);
                cuda_convert_f32_to_f16(l.weights_gpu, l.nweights, l.weights_gpu16);
            }
            else if (l.type == CRNN && l.input_layer->weights_gpu && l.input_layer->weights_gpu16) {
                assert((l.input_layer->c*l.input_layer->n*l.input_layer->size*l.input_layer->size) > 0);
                cuda_convert_f32_to_f16(l.input_layer->weights_gpu, l.input_layer->nweights, l.input_layer->weights_gpu16);
                cuda_convert_f32_to_f16(l.self_layer->weights_gpu, l.self_layer->nweights, l.self_layer->weights_gpu16);
                cuda_convert_f32_to_f16(l.output_layer->weights_gpu, l.output_layer->nweights, l.output_layer->weights_gpu16);
            }
            else if (l.type == CONV_LSTM && l.wf->weights_gpu && l.wf->weights_gpu16) {
                assert((l.wf->c * l.wf->n * l.wf->size * l.wf->size) > 0);
                if (l.peephole) {
                    cuda_convert_f32_to_f16(l.vf->weights_gpu, l.vf->nweights, l.vf->weights_gpu16);
                    cuda_convert_f32_to_f16(l.vi->weights_gpu, l.vi->nweights, l.vi->weights_gpu16);
                    cuda_convert_f32_to_f16(l.vo->weights_gpu, l.vo->nweights, l.vo->weights_gpu16);
                }
                cuda_convert_f32_to_f16(l.wf->weights_gpu, l.wf->nweights, l.wf->weights_gpu16);
                cuda_convert_f32_to_f16(l.wi->weights_gpu, l.wi->nweights, l.wi->weights_gpu16);
                cuda_convert_f32_to_f16(l.wg->weights_gpu, l.wg->nweights, l.wg->weights_gpu16);
                cuda_convert_f32_to_f16(l.wo->weights_gpu, l.wo->nweights, l.wo->weights_gpu16);
                cuda_convert_f32_to_f16(l.uf->weights_gpu, l.uf->nweights, l.uf->weights_gpu16);
                cuda_convert_f32_to_f16(l.ui->weights_gpu, l.ui->nweights, l.ui->weights_gpu16);
                cuda_convert_f32_to_f16(l.ug->weights_gpu, l.ug->nweights, l.ug->weights_gpu16);
                cuda_convert_f32_to_f16(l.uo->weights_gpu, l.uo->nweights, l.uo->weights_gpu16);
            }
        }
    }
#endif
    forward_network_gpu(net, state);
    //hipStreamSynchronize(get_cuda_stream());
    backward_network_gpu(net, state);
}

float train_network_datum_gpu(network net, float *x, float *y)
{
    *net.seen += net.batch;
    forward_backward_network_gpu(net, x, y);
    float error = get_network_cost(net);
    //if (((*net.seen) / net.batch) % net.subdivisions == 0) update_network_gpu(net);
    const int sequence = get_sequence_value(net);
    if (((*net.seen) / net.batch) % (net.subdivisions*sequence) == 0) update_network_gpu(net);

    return error;
}

typedef struct {
    network net;
    data d;
    float *err;
} train_args;

void *train_thread(void *ptr)
{
    train_args args = *(train_args*)ptr;
    free(ptr);
    cuda_set_device(args.net.gpu_index);
    *args.err = train_network(args.net, args.d);
    return 0;
}

pthread_t train_network_in_thread(network net, data d, float *err)
{
    pthread_t thread;
    train_args *ptr = (train_args *)calloc(1, sizeof(train_args));
    ptr->net = net;
    ptr->d = d;
    ptr->err = err;
    if(pthread_create(&thread, 0, train_thread, ptr)) error("Thread creation failed");
    return thread;
}

void pull_updates(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
        if(l.scale_updates) cuda_pull_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
}

void push_updates(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
        if(l.scale_updates) cuda_push_array(l.scale_updates_gpu, l.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
    }
}

void update_layer(layer l, network net)
{
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    l.t = get_current_batch(net);
    if(l.update_gpu){
        l.update_gpu(l, update_batch, rate, net.momentum, net.decay);
    }
}

void merge_weights(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.nweights, 1, l.weights, 1, base.weights, 1);
        if (l.scales) {
            axpy_cpu(l.n, 1, l.scales, 1, base.scales, 1);
        }
    } else if(l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.outputs*l.inputs, 1, l.weights, 1, base.weights, 1);
    }
}

void scale_weights(layer l, float s)
{
    if (l.type == CONVOLUTIONAL) {
        scal_cpu(l.n, s, l.biases, 1);
        scal_cpu(l.nweights, s, l.weights, 1);
        if (l.scales) {
            scal_cpu(l.n, s, l.scales, 1);
        }
    } else if(l.type == CONNECTED) {
        scal_cpu(l.outputs, s, l.biases, 1);
        scal_cpu(l.outputs*l.inputs, s, l.weights, 1);
    }
}


void pull_weights(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_pull_array(l.biases_gpu, l.biases, l.n);
        cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
        if(l.scales) cuda_pull_array(l.scales_gpu, l.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_pull_array(l.biases_gpu, l.biases, l.outputs);
        cuda_pull_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
    }
}

void push_weights(layer l)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.biases_gpu, l.biases, l.n);
        cuda_push_array(l.weights_gpu, l.weights, l.nweights);
        if(l.scales) cuda_push_array(l.scales_gpu, l.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.biases_gpu, l.biases, l.outputs);
        cuda_push_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
    }
}

void distribute_weights(layer l, layer base)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.biases_gpu, base.biases, l.n);
        cuda_push_array(l.weights_gpu, base.weights, l.nweights);
        if(base.scales) cuda_push_array(l.scales_gpu, base.scales, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.biases_gpu, base.biases, l.outputs);
        cuda_push_array(l.weights_gpu, base.weights, l.outputs*l.inputs);
    }
}


void merge_updates(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.nweights, 1, l.weight_updates, 1, base.weight_updates, 1);
        if (l.scale_updates) {
            axpy_cpu(l.n, 1, l.scale_updates, 1, base.scale_updates, 1);
        }
    } else if(l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.outputs*l.inputs, 1, l.weight_updates, 1, base.weight_updates, 1);
    }
}

void distribute_updates(layer l, layer base)
{
    if(l.type == CONVOLUTIONAL){
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.nweights);
        if(base.scale_updates) cuda_push_array(l.scale_updates_gpu, base.scale_updates, l.n);
    } else if(l.type == CONNECTED){
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.outputs*l.inputs);
    }
}

void sync_layer(network *nets, int n, int j)
{
    //printf("Syncing layer %d\n", j);
    int i;
    network net = nets[0];
    layer base = net.layers[j];
    cuda_set_device(net.gpu_index);
    pull_weights(base);
    for (i = 1; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        pull_weights(l);
        merge_weights(l, base);
    }
    scale_weights(base, 1./n);
    for (i = 0; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        distribute_weights(l, base);
    }
    //printf("Done syncing layer %d\n", j);
}

typedef struct{
    network *nets;
    int n;
    int j;
} sync_args;

void *sync_layer_thread(void *ptr)
{
    sync_args args = *(sync_args*)ptr;
    sync_layer(args.nets, args.n, args.j);
    free(ptr);
    return 0;
}

pthread_t sync_layer_in_thread(network *nets, int n, int j)
{
    pthread_t thread;
    sync_args *ptr = (sync_args *)calloc(1, sizeof(sync_args));
    ptr->nets = nets;
    ptr->n = n;
    ptr->j = j;
    if(pthread_create(&thread, 0, sync_layer_thread, ptr)) error("Thread creation failed");
    return thread;
}

void sync_nets(network *nets, int n, int interval)
{
    int j;
    int layers = nets[0].n;
    pthread_t *threads = (pthread_t *) calloc(layers, sizeof(pthread_t));

    *nets[0].seen += interval * (n-1) * nets[0].batch * nets[0].subdivisions;
    for (j = 0; j < n; ++j){
        *nets[j].seen = *nets[0].seen;
    }
    for (j = 0; j < layers; ++j) {
        threads[j] = sync_layer_in_thread(nets, n, j);
    }
    for (j = 0; j < layers; ++j) {
        pthread_join(threads[j], 0);
    }
    free(threads);
}

float train_networks(network *nets, int n, data d, int interval)
{
    int i;
#ifdef _DEBUG
    int batch = nets[0].batch;
    int subdivisions = nets[0].subdivisions;
    assert(batch * subdivisions * n == d.X.rows);
#endif
    pthread_t *threads = (pthread_t *) calloc(n, sizeof(pthread_t));
    float *errors = (float *) calloc(n, sizeof(float));

    float sum = 0;
    for(i = 0; i < n; ++i){
        data p = get_data_part(d, i, n);
        threads[i] = train_network_in_thread(nets[i], p, errors + i);
    }
    for(i = 0; i < n; ++i){
        pthread_join(threads[i], 0);
        //printf("%f\n", errors[i]);
        sum += errors[i];
    }
    //hipDeviceSynchronize();
    if (get_current_batch(nets[0]) % interval == 0) {
        printf("Syncing... ");
        fflush(stdout);
        sync_nets(nets, n, interval);
        printf("Done!\n");
    }
    //hipDeviceSynchronize();
    free(threads);
    free(errors);
    return (float)sum/(n);
}

float *get_network_output_layer_gpu(network net, int i)
{
    double _time = get_time_point();
    layer l = net.layers[i];
    if(l.type != REGION){
        printf("l.type is %s\n",get_layer_string(l.type));
        printf("test_extern_arr : %d\n",test_extern_arr[i]);
        if(test_extern_arr[i] == 1){//from gpu
            printf("pulled from gpu.\n");
            cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);
        }
    }

    printf("end of get_net_output, time is %8.5f millisec\n",((double)get_time_point() - _time)/1000);
    return l.output;
}

float *get_network_output_gpu(network net)
{
    int i;
    for(i = net.n-1; i > 0; --i) if(net.layers[i].type != COST) break;
    printf("target layer i is %d.\n",i);
    return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(network net, float *input)
{
    int* res_arr;           // change the scope of memory according to resource allocation.
    float* temp_ptr[net.n]; // temporary pointers for hipMalloc or malloc memories.
    double _time_cp;        // gpu_memcpy_timer.
    int i;

    double _time = get_time_point();
    if (net.gpu_index != cuda_get_device())
        cuda_set_device(net.gpu_index);
    int size = get_network_input_size(net) * net.batch;
    network_state state;
    state.index = 0;
    state.net = net;
    //state.input = cuda_make_array(input, size);   // memory will be allocated in the parse_network_cfg_custom() 
    
    res_arr = test_extern_arr;

    if (res_arr[0] == 0){//first network runs on cpu.
        memcpy(net.input_pinned_cpu, input, size*sizeof(float));
        state.input = net.input_pinned_cpu;
    //      printf("this is input%d\n",*state.input);
    }
    else{//first network runs on gpu.
        state.input = net.input_state_gpu;
        _time_cp = get_time_point();//init timer.
        memcpy(net.input_pinned_cpu, input, size * sizeof(float));
        cuda_push_array(state.input, net.input_pinned_cpu, size);
    }
    state.truth = 0;
    state.train = 0;
    state.delta = 0;

    //allocate unified cuda memories.
    printf("start of unified memory reallocation\n");
    for(i = 0; i < net.n; ++i){
        if((res_arr[i] != res_arr[i+1]) || (i==8) || (i==13) || (i==19)){//computation resource change || route layer target.
            layer *lptr = &(net.layers[i]);
            if(res_arr[i] == 0){//if prev resource was CPU
                temp_ptr[i] = lptr->output;
                lptr->output = cuda_make_array_global(lptr->output,lptr->batch * lptr->outputs);
            }
            else{//if prev resource was GPU
                temp_ptr[i] = lptr->output_gpu;
                lptr->output_gpu = cuda_make_array_global(lptr->output,lptr->batch * lptr->outputs);
            }
        }
    }
    printf("end of unified memory reallocation\n");
    //!allocated.

    forward_network_gpu(net, state);
    float *out = get_network_output_gpu(net);

    //free cuda memories and return original memory pointer.
    printf("start of returning memory reallocation\n");
    for(i=0; i<net.n; ++i){
        if(res_arr[i] != res_arr[i+1]){
            layer *lptr = &(net.layers[i]);
            if(res_arr[i] == 0){
                cuda_free(lptr->output);
                lptr->output = temp_ptr[i];
            }
            else{
                cuda_free(lptr->output_gpu);
                lptr->output_gpu = temp_ptr[i];
            }
        }
    }
    printf("end of returning memory reallocation\n");
    //!freed.

    //cuda_free(state.input);   // will be freed in the free_network()
    return out;
}


////////////// GPU ACCESSING MANIGNING //////////////
void swap(int *xp, int *yp)
{
    int temp = *xp;
    *xp = *yp;
    *yp = temp;
}

void bubbleSort(int arr[], int n)
{
   int i, j;
   for (i = 0; i < n-1; i++)      
 
       // Last i elements are already in place   
       for (j = 0; j < n-i-1; j++) 
           if (arr[j] < arr[j+1])
              swap(&arr[j], &arr[j+1]);
}

void enqueue(int* q, int val)
{
  for (int i=0; i<N; i++){
	if (q[i] == 0){
	q[i] = val;
	break;
	}
  }  
}

int dequeue(int* q)
{
	/* sort */
	bubbleSort(q, N);	

	int tmp =  q[0];
	for (int i=0; q[i]>0; i++){		
	q[i] = q[i+1];
	}

	return tmp;
}
